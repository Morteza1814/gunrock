#include "hip/hip_runtime.h"
#include <gunrock/algorithms/spmv.hxx>
#include <gunrock/algorithms/generate/random.hxx>
#include "spmv_cpu.hxx"

using namespace gunrock;
using namespace memory;

void test_spmv(int num_arguments, char** argument_array) {
  if (num_arguments != 5) {
    std::cerr << "usage: ./bin/<program-name> -m filename.mtx -n num_runs" << std::endl;
    exit(1);
  }

  // --
  // Define types
  // Specify the types that will be used for
  // - vertex ids (vertex_t)
  // - edge offsets (edge_t)
  // - edge weights (weight_t)

  using vertex_t = int;
  using edge_t = int;
  using weight_t = float;
  using csr_t =
      format::csr_t<memory_space_t::device, vertex_t, edge_t, weight_t>;

  // --
  // IO

  // Filename to be read
  std::string filename = argument_array[2];
  uint32_t num_runs = std::stoi(argument_array[4]);
  // Load the matrix-market dataset into csr format.
  // See `format` to see other supported formats.
  io::matrix_market_t<vertex_t, edge_t, weight_t> mm;
  auto [properties, coo] = mm.load(filename);

  csr_t csr;
  csr.from_coo(coo);

  // --
  // Build graph

  // Convert the dataset you loaded into an `essentials` graph.
  // `memory_space_t::device` -> the graph will be created on the GPU.
  auto G = graph::build<memory_space_t::device>(properties, csr);

  std::vector<float> run_times;
  // --
  // Params and memory allocation
  vertex_t n_vertices = G.get_number_of_vertices();
  thrust::device_vector<weight_t> x(n_vertices);
  thrust::device_vector<weight_t> y(n_vertices);

  for (int i = 0; i < num_runs; i++) {
    gunrock::generate::random::uniform_distribution(x);

    // --
    // GPU Run
    run_times.push_back(gunrock::spmv::run(G, x.data().get(), y.data().get()));
  }

  // --
  // CPU Run

  thrust::host_vector<weight_t> y_h(n_vertices);
  float cpu_elapsed = spmv_cpu::run(csr, x, y_h);

  // --
  // Log + Validate
  int n_errors = util::compare(
      y.data().get(), y_h.data(), n_vertices,
      [=](const weight_t a, const weight_t b) {
        // TODO: needs better accuracy.
        return std::abs(a - b) > 1e-2;
      },
      true);

  gunrock::print::head(y, 40, "GPU y-vector");
  gunrock::print::head(y_h, 40, "CPU y-vector");

  // std::cout << "GPU Elapsed Time : " << gpu_elapsed << " (ms)" << std::endl;
  std::cout << "CPU Elapsed Time : " << cpu_elapsed << " (ms)" << std::endl;
  std::cout << "Number of errors : " << n_errors << std::endl;

  float total_time = 0;
  for (int i = 0; i < num_runs; i++) {
    std::cout << "Run " << i << " Exec Time: " << run_times[i] << " (ms)" << std::endl;
    //get the average time
    total_time += run_times[i];
  }
  std::cout << "Average GPU Elapsed Time : " << (float)(total_time/num_runs) << " (ms)"
            << std::endl;
}

// Main method, wrapping test function
int main(int argc, char** argv) {
  test_spmv(argc, argv);
}